#include "hip/hip_runtime.h"
#include "triangle.h"
#include <helper_functions.h> // helper functions for SDK examples
#include ""
#include <hip/hip_runtime_api.h>
#include "iostream"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


using namespace std;


__global__ void PreComputeTriangle_GPU(unsigned int* m_vVertexMap,double* m_mFirstMatrix,unsigned int row,unsigned int col,Triangle_GPU* m_vTriangles,Vertex_GPU* m_Vertex,unsigned int nVerts){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx>=nVerts)return;
	
	Vertex_GPU & v = m_Vertex[idx];
	
	
	for (int i = 0; i < v.nTriangle; i++){
		Triangle_GPU & t = m_vTriangles[v.Triangles[i]];
		int j = 0;
		//printf("#%d v %d %d %d",idx,);
		for (j; j < 3; j++){
			if (idx == t.nVert[j])
				break;
		}
		if (j == 3){ printf("vertex error idx %d nVert %d %d %d\n", idx, t.nVert[0], t.nVert[1], t.nVert[2]); return; }
		
		int n0x = 2 * m_vVertexMap[t.nVert[j]];
		int n0y = n0x + 1;
		int n1x = 2 * m_vVertexMap[t.nVert[(j + 1) % 3]];
		int n1y = n1x + 1;
		int n2x = 2 * m_vVertexMap[t.nVert[(j + 2) % 3]];
		int n2y = n2x + 1;

		float x0 = t.X[j];
		float y0 = t.Y[j];
		float x1 = t.X[(j + 2) % 3];
		float y1 = t.Y[(j + 2) % 3];
		float x2 = t.X[(j + 1) % 3];
		float y2 = t.Y[(j + 1) % 3];

		//for n0

		m_mFirstMatrix[n0x*row + n0x] += 1 - 2 * x0 + x0*x0 + y0*y0;
		m_mFirstMatrix[n0x*row + n1x] += 2 * x0 - 2 * x0*x0 - 2 * y0*y0;		//m_mFirstMatrix[n1x][n0x] += 2*x - 2*x*x - 2*y*y;
		m_mFirstMatrix[n0x*row + n1y] += 2 * y0;						//m_mFirstMatrix[n1y][n0x] += 2*y;
		m_mFirstMatrix[n0x*row + n2x] += -2 + 2 * x0;					//m_mFirstMatrix[n2x][n0x] += -2 + 2*x;
		m_mFirstMatrix[n0x*row + n2y] += -2 * y0;

		m_mFirstMatrix[n0y*row + n0y] += 1 - 2 * x0 + x0*x0 + y0*y0;
		m_mFirstMatrix[n0y*row + n1x] += -2 * y0;						//m_mFirstMatrix[n1x][n0y] += -2*y;
		m_mFirstMatrix[n0y*row + n1y] += 2 * x0 - 2 * x0*x0 - 2 * y0*y0;		//m_mFirstMatrix[n1y][n0y] += 2*x - 2*x*x - 2*y*y;
		m_mFirstMatrix[n0y*row + n2x] += 2 * y0;						//m_mFirstMatrix[n2x][n0y] += 2*y;
		m_mFirstMatrix[n0y*row + n2y] += -2 + 2 * x0;

		//for n1
		// n1x,n?? elems
		m_mFirstMatrix[n0x*row + n0x] += x1*x1 + y1*y1;
		m_mFirstMatrix[n0x*row + n1x] += -2 * x1;						//m_mFirstMatrix[n2x][n1x] += -2*x;
		m_mFirstMatrix[n0x*row + n1y] += 2 * y1;						//m_mFirstMatrix[n2y][n1x] += 2*y;


		//n1y,n?? elems
		m_mFirstMatrix[n0y*row + n0y] += x1*x1 + y1*y1;
		m_mFirstMatrix[n0y*row + n1x] += -2 * y1;						//m_mFirstMatrix[n2x][n1y] += -2*y;
		m_mFirstMatrix[n0y*row + n1y] += -2 * x1;						//m_mFirstMatrix[n2y][n1y] += -2*x;

		//for n2
		// final 2 elems
		m_mFirstMatrix[n0x*row + n0x] += 1;
		m_mFirstMatrix[n0y*row + n0y] += 1;
		

	}


	/*double fTriSumErr = 0;
		for ( int j = 0; j < 3; ++j ) {
			double fTriErr = 0;

			int n0x = 2 * m_vVertexMap[ t.nVert[j] ];
			int n0y = n0x + 1;
			int n1x = 2 * m_vVertexMap[ t.nVert[(j+1)%3] ];
			int n1y = n1x + 1;
			int n2x = 2 * m_vVertexMap[ t.nVert[(j+2)%3] ];
			int n2y = n2x + 1;
			
			/*n0xA[threadIdx.x][j] = 2 * m_vVertexMap[t.nVert[j]];
			//printf("~~~~~~%d", n0xA[threadIdx.x][j]);
			n0yA[threadIdx.x][j] = n0xA[threadIdx.x][j] + 1;
			n1xA[threadIdx.x][j] = 2 * m_vVertexMap[t.nVert[(j + 1) % 3]];
			n1yA[threadIdx.x][j] = n1xA[threadIdx.x][j] + 1;
			n2xA[threadIdx.x][j] = 2 * m_vVertexMap[t.nVert[(j + 2) % 3]];
			n2yA[threadIdx.x][j] = n2yA[threadIdx.x][j] + 1;
			printf("~");


			float x = t.X[j];
			float y = t.Y[j];
			if ((idx == 0||idx==1)&&j==0)
				printf("test GPU value %d,%d n0x %d n0y %d x %d y%d\n",idx,j, n0x, n0y, x, y);
			


			m_mFirstMatrix[n0x*row+n0x] += 1 - 2*x + x*x + y*y;
			m_mFirstMatrix[n0x*row+n1x] += 2*x - 2*x*x - 2*y*y;		//m_mFirstMatrix[n1x][n0x] += 2*x - 2*x*x - 2*y*y;
			m_mFirstMatrix[n0x*row+n1y] += 2*y;						//m_mFirstMatrix[n1y][n0x] += 2*y;
			m_mFirstMatrix[n0x*row+n2x] += -2 + 2*x;					//m_mFirstMatrix[n2x][n0x] += -2 + 2*x;
			m_mFirstMatrix[n0x*row+n2y] += -2 * y;						//m_mFirstMatrix[n2y][n0x] += -2 * y;



			// n0y,n?? elems
			m_mFirstMatrix[n0y*row+n0y] += 1 - 2*x + x*x + y*y;
			m_mFirstMatrix[n0y*row+n1x] += -2*y;						//m_mFirstMatrix[n1x][n0y] += -2*y;
			m_mFirstMatrix[n0y*row+n1y] += 2*x - 2*x*x - 2*y*y;		//m_mFirstMatrix[n1y][n0y] += 2*x - 2*x*x - 2*y*y;
			m_mFirstMatrix[n0y*row+n2x] += 2*y;						//m_mFirstMatrix[n2x][n0y] += 2*y;
			m_mFirstMatrix[n0y*row+n2y] += -2 + 2*x;					//m_mFirstMatrix[n2y][n0y] += -2 + 2*x;



			// n1x,n?? elems
			m_mFirstMatrix[n1x*row+n1x] += x*x + y*y;
			m_mFirstMatrix[n1x*row+n2x] += -2*x;						//m_mFirstMatrix[n2x][n1x] += -2*x;
			m_mFirstMatrix[n1x*row+n2y] += 2*y;						//m_mFirstMatrix[n2y][n1x] += 2*y;


			//n1y,n?? elems
			m_mFirstMatrix[n1y*row+n1y] += x*x + y*y;
			m_mFirstMatrix[n1y*row+n2x] += -2*y;						//m_mFirstMatrix[n2x][n1y] += -2*y;
			m_mFirstMatrix[n1y*row+n2y] += -2*x;						//m_mFirstMatrix[n2y][n1y] += -2*x;



			// final 2 elems
			m_mFirstMatrix[n2x*row+n2x] += 1;
			m_mFirstMatrix[n2y*row+n2y] += 1;

		}
		*/


		//_RMSInfo("  Total Error: %f\n", fTriSumErr);

		if (idx == 1)
			printf("output test GPU %lf", m_mFirstMatrix[0]);
}

void PreComputeTriangle(unsigned int*m_vVertexMap_GPU, double* m_mFirstMatrix, unsigned int row, unsigned int col, Triangle_GPU* m_vTriangles,Vertex_GPU* m_Vertex, unsigned int nVerts){
	PreComputeTriangle_GPU<<< nVerts/64+1,64 >>>(m_vVertexMap_GPU, m_mFirstMatrix, row, col, m_vTriangles, m_Vertex, nVerts);
}